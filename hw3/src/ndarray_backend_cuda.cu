#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256

#define TILE 4
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  uint32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<uint32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides




__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN YOUR SOLUTION
  if (gid < size) {
    uint32_t idx = offset;
    size_t v = gid;
    size_t p = 0;
    for (size_t i = 0; i < shape.size; ++i) {
      p = v % shape.data[shape.size - i - 1];
      v = v / shape.data[shape.size - i - 1];
      idx += p * strides.data[strides.size - i - 1];
    }
    out[gid] = a[idx];
  }
  /// END YOUR SOLUTION
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
             std::vector<uint32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}


__global__ void EwiseSetItemKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    uint32_t idx = offset;
    size_t v = gid;
    size_t p = 0;
    for (size_t i = 0; i < shape.size; ++i) {
      p = v % shape.data[shape.size - i - 1];
      v = v / shape.data[shape.size - i - 1];
      idx += p * strides.data[strides.size - i - 1];
    }
    out[idx] = a[gid];
  }        
}

void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
                  std::vector<uint32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(a.size);
  EwiseSetItemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, VecToCuda(shape),
                                              VecToCuda(strides), offset);
  
  /// END YOUR SOLUTION
}

__global__ void ScalarSetItemKernel(scalar_t val, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    uint32_t idx = offset;
    size_t v = gid;
    size_t p = 0;
    for (size_t i = 0; i < shape.size; ++i) {
      p = v % shape.data[shape.size - i - 1];
      v = v / shape.data[shape.size - i - 1];
      idx += p * strides.data[strides.size - i - 1];
    }
    out[idx] = val;
  }        
}

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<uint32_t> shape,
                   std::vector<uint32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(size);
  ScalarSetItemKernel<<<dim.grid, dim.block>>>(val, out->ptr, size, VecToCuda(shape),
                                              VecToCuda(strides), offset);
  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */

/// BEGIN YOUR SOLUTION
#define EwiseKernel(name, op) \
  __global__ void EwiseKernel##name(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) { \
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;                                                \
    if (gid < size) out[gid] = a[gid] op b[gid];                                                       \
  }                                                                                                    \

#define EWISEOP(name) \
  void Ewise##name(const CudaArray& a, const CudaArray& b, CudaArray* out) {      \
    CudaDims dim = CudaOneDim(out->size);                                         \
    EwiseKernel##name<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);\
  }                                                                               \

EwiseKernel(Mul, *)
EwiseKernel(Div, /)
EWISEOP(Mul)
EWISEOP(Div)

#define ScalarKernel(name, op) \
  __global__ void ScalarKernel##name(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) { \
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;                                            \
    if (gid < size) out[gid] = a[gid] op val;                                                      \
  }                                                                                                \

#define SCALAROP(name) \
  void Scalar##name(const CudaArray& a, scalar_t val, CudaArray* out) {           \
    CudaDims dim = CudaOneDim(out->size);                                         \
    ScalarKernel##name<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size); \
  }                                                                               \

ScalarKernel(Mul, *)
ScalarKernel(Div, /)
SCALAROP(Mul)
SCALAROP(Div)

__global__ void ScalarPowerKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = pow(a[gid], val);
}

void ScalarPower(const CudaArray& a, scalar_t val, CudaArray* out) { 
  CudaDims dim = CudaOneDim(out->size);
  ScalarPowerKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseMaximumKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x; 
  if (gid < size) out[gid] = (a[gid] > b[gid]) ? a[gid] : b[gid]; 
}

void EwiseMaximum(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size); 
  EwiseMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMaximumKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = (a[gid] > val) ? a[gid] : val;
}

void ScalarMaximum(const CudaArray& a, scalar_t val, CudaArray* out)  { 
  CudaDims dim = CudaOneDim(out->size);
  ScalarMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

#define EwiseCompKernel(name, op) \
  __global__ void EwiseKernel##name(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) { \
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;                                                \
    if (gid < size) out[gid] = (a[gid] op b[gid]) ? 1: 0;                                              \
  }                                                                                                    \

#define EWISECOMP(name) \
  void Ewise##name(const CudaArray& a, const CudaArray& b, CudaArray* out) {          \
    CudaDims dim = CudaOneDim(out->size);                                             \
    EwiseKernel##name<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);    \
  }                                                                                   \

EwiseCompKernel(Eq, ==)
EwiseCompKernel(Ge, >=)
EWISECOMP(Eq)
EWISECOMP(Ge)

#define ScalarCompKernel(name, op) \
  __global__ void ScalarKernel##name(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) { \
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;                                            \
    if (gid < size) out[gid] = (a[gid] op val) ? 1: 0;                                             \
  }                                                                                                \

#define SCALARCOMP(name) \
  void Scalar##name(const CudaArray& a, scalar_t val, CudaArray* out) {           \
    CudaDims dim = CudaOneDim(out->size);                                         \
    ScalarKernel##name<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size); \
  }                                                                               \


ScalarCompKernel(Eq, ==)
ScalarCompKernel(Ge, >=)
SCALARCOMP(Eq)
SCALARCOMP(Ge)


#define EwiseCalKernel(name, op) \
  __global__ void EwiseKernel##name(const scalar_t* a, scalar_t* out, size_t size) { \
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x;                             \
    if (gid < size) out[gid] = op(a[gid]);                                          \
  }                                                                                 \

#define EWISECAL(name) \
  void Ewise##name(const CudaArray& a, CudaArray* out) {                      \
    CudaDims dim = CudaOneDim(out->size);                                     \
    EwiseKernel##name<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);   \
  }                                                                           \


EwiseCalKernel(Log, std::log)
EwiseCalKernel(Exp, std::exp)
EwiseCalKernel(Tanh, std::tanh)
EWISECAL(Log)
EWISECAL(Exp)
EWISECAL(Tanh)

/// END YOUR SOLUTION

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////
 
// A  m*n
// B  n*p
// C  m*p
template <size_t BLOCK_SIZE, size_t SHARED_LEN>
__global__ void MatmulKernel(const scalar_t* A, const scalar_t* B, scalar_t* C, uint32_t M, uint32_t N, uint32_t P) {
  const size_t V = BLOCK_SIZE;
  const size_t L = SHARED_LEN;
  __shared__ float sA[V][L];
  __shared__ float sB[V][L];
  scalar_t a[V];
  scalar_t b[V];
  scalar_t c = 0;

  size_t by = blockIdx.y;
  size_t bx = blockIdx.x;
  size_t tx = threadIdx.x;
  size_t ty = threadIdx.y;
  size_t cx = bx * V + tx;
  size_t cy = by * V + ty;

  for (int k = 0; k < N; k += L) {
    __syncthreads();
  // load A from global mem to shared mem
  for (int j = 0; j < L; j += V) {
    size_t gid = (by*V + ty) * N + (tx + j + k);
    size_t y = ty;
    size_t x = j + tx;
    if (gid < M*N && (ty + by * V) < M && (tx + j + k) < N) {
      sA[y][x] = A[gid];
    } else {
      sA[y][x] = 0;
    }
    
  }
  // load B.T from global mem to shared mem
  for (int j = 0; j < L; j += V) {
    size_t gid = (bx*V + tx) + (ty + j + k) * P; 
    size_t y = tx;
    size_t x = j + ty;
    if (gid < N*P && (ty + j + k) < N && (tx + bx *V) < P) {
      sB[y][x] = B[gid];
    } else {
      sB[y][x] = 0;
    }
  }
  __syncthreads();

  
  if (cx < P && cy < M) {
    for (size_t j = 0; j*V < L; ++j) {
      // load from shared mem to register
      for (size_t cnt = 0; cnt < V; ++cnt) {
        a[cnt] = sA[ty][j * V + cnt];
      }
      for (size_t cnt = 0; cnt < V; ++cnt) {
        b[cnt] = sB[tx][j * V + cnt];
      }
      // calculate
      for (size_t i = 0; i < V; ++i) {
        c += a[i] * b[i];
      }
    }
    
  }
  }

  // load mem to global c
  if (cx < P && cy < M) {
    C[cy*P + cx] = c;
  }
  
}


void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN YOUR SOLUTION

  const size_t BS = 4;
  const size_t SL = 32;
  dim3 block(BS, BS, 1);
	dim3 grid((P + BS - 1) / BS, (M + BS - 1) / BS, 1);
  MatmulKernel<BS, SL> <<<grid, block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  
  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////

__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t* out, size_t reduce_size) {
  size_t index = threadIdx.x * reduce_size; 
  scalar_t ret = a[index];
  for (size_t i = index + 1; i < index + reduce_size; ++i) {
    ret = (ret > a[i]) ? ret : a[i];
  }
  out[threadIdx.x] = ret;
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  ReduceMaxKernel<<<1, out->size>>>(a.ptr, out->ptr, reduce_size);
  /// END YOUR SOLUTION
}


__global__ void ReduceSumKernel(const scalar_t* a, scalar_t* out, size_t reduce_size) {
  size_t index = threadIdx.x * reduce_size; 
  scalar_t ret = a[index];
  for (size_t i = index + 1; i < index + reduce_size; ++i) {
    ret += a[i];
  }
  out[threadIdx.x] = ret;
}

void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  ReduceSumKernel<<<1, out->size>>>(a.ptr, out->ptr, reduce_size);
  /// END YOUR SOLUTION
}

}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}
